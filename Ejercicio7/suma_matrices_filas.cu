
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <time.h>
const int N=10000;
const int NBLOCK=1024;

__global__ void MatAdd( float *A, float *B, float *C, int N)
{
/*int j = blockIdx.x * blockDim.x + threadIdx.x;  // Compute row index
int i = blockIdx.y * blockDim.y + threadIdx.y;  // Compute column index
int index=i*N+j; // Compute global 1D index
if (i < N && j < N)
	C[index] = A[index] + B[index]; // Compute C element*/
int idHebra = blockIdx.x * blockDim.x + threadIdx.x; 
if (idHebra < N){
	//int inicio = idHebra * N;
	for (int i = 0; i < N; i++){
		int index = idHebra * N + i;
		C[index] = A[index] + B[index];
	}
}

}

int main()
{
int i;
const int NN=N*N;
/* pointers to host memory */
/* Allocate arrays A, B and C on host*/
float * A = (float*) malloc(NN*sizeof(float));
float * B = (float*) malloc(NN*sizeof(float));
float * C = (float*) malloc(NN*sizeof(float));

/* pointers to device memory */
float *A_d, *B_d, *C_d;
/* Allocate arrays a_d, b_d and c_d on device*/
hipMalloc ((void **) &A_d, sizeof(float)*NN);
hipMalloc ((void **) &B_d, sizeof(float)*NN);
hipMalloc ((void **) &C_d, sizeof(float)*NN);

/* Initialize arrays a and b */
for (i=0; i<NN;i++)
{
  A[i]= (float) 2;
  B[i]= (float) 2;
}


clock_t begin_time = clock();
/* Copy data from host memory to device memory */
hipMemcpy(A_d, A, sizeof(float)*NN, hipMemcpyHostToDevice);
hipMemcpy(B_d, B, sizeof(float)*NN, hipMemcpyHostToDevice);

/* Compute the execution configuration */
/*dim3 threadsPerBlock (16, 16);
dim3 numBlocks( ceil ((float)(N)/threadsPerBlock.x), ceil ((float)(N)/threadsPerBlock.y) );*/
MatAdd <<<ceil((float)N/NBLOCK), NBLOCK>>> (A_d, B_d, C_d, N);


/* Copy data from deveice memory to host memory */
hipMemcpy(C, C_d, sizeof(float)*NN, hipMemcpyDeviceToHost);

double Tgpu = float(clock() - begin_time) / CLOCKS_PER_SEC;
printf(" El tiempo consumido es de %f segundos", Tgpu);
/* Print c */
/*for (i=0; i<NN;i++)
  printf(" c[%d]=%f\n",i,C[i]);*/

/* Free the memory */
free(A); free(B); free(C);
hipFree(A_d); hipFree(B_d);hipFree(C_d);



}
