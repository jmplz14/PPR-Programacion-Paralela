#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string.h>
#include <time.h>


// CUDA runtime
//#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>




using namespace std;

//**************************************************************************
/*double cpuSecond()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
}*/

//**************************************************************************
__global__ void transformacionSinCompartida(float * A, float * B, float * C, int N) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < N) {
		int K = N / i;
		int posInicio = K * blockDim.x;
		float suma = 0;
		for (int j = 0; j < blockDim.x; j++) {
			int posActual = posInicio + j;
			float valorA = A[posActual] * i;
			if ( ( (int)ceil(valorA) % 2) == 0) {
				suma += valorA + B[posActual];
			}
			else {
				suma += valorA - B[posActual];
			}
		}
		C[i] = suma;
	}
	
}

__global__ void transformacionConCompartida(float * A, float * B, float * C, int N) {
	extern __shared__ float sdata[];
	int tid = threadIdx.x;
	int i = tid + blockDim.x * blockIdx.x;
	
	float *sdata_A = sdata; 
	float *sdata_B = sdata + blockDim.x; 

	sdata_A[tid] = A[i];
	sdata_B[tid] = B[i];

	__syncthreads();

	if (i < N) {
		int K = N / i;
		int posInicio = K * blockDim.x;
		float suma = 0;

		for (int j = 0; j < blockDim.x; j++) {
			int posActual = posInicio + j;
			float valorA = sdata_A[posActual] * i;

			if (((int)ceil(valorA) % 2) == 0) {
				suma += valorA + sdata_B[posActual];
			}
			else {
				suma += valorA - sdata_B[posActual];
			}
		}
		C[i] = suma;
	}

}








int main(int argc, char *argv[]) {

	int blocksize, NBlocks;
	if (argc != 3)
	{
		cout << "Uso: transformacion Num_bloques Tam_bloque  " << endl;
		return(0);
	}
	else
	{
		NBlocks = atoi(argv[1]);
		blocksize = atoi(argv[2]);
	}

	const int   N = blocksize * NBlocks;

	float *A = new float[N];
	float *B = new float[N];
	float *C = new float[N];

	int devID;
	hipError_t err;
	err = hipGetDevice(&devID);
	if (err != hipSuccess) {
		cout << "ERRORRR" << endl;
	}
	
	int size = N * sizeof(float);
	float * d_A = NULL, *d_B = NULL, *d_C = NULL;

	err = hipMalloc((void **)&d_A, size);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA A" << endl;
	}

	err = hipMalloc((void **)&d_B, size);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA B" << endl;
	}

	err = hipMalloc((void **)&d_C, size);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA B" << endl;
	}

	err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "ERROR COPIA A" << endl;
	}

	err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "ERROR COPIA B" << endl;
	}

	err = hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "ERROR COPIA C" << endl;
	}

	for (int i = 0; i < N; i++)
	{
		A[i] = (float)(1 - (i % 100)*0.001);
		B[i] = (float)(0.5 + (i % 10) *0.1);
	}
	
	int threadsPerBlock = blocksize;
	int blocksPerGrid = ceil((float)N/threadsPerBlock);

	double  t1 = clock();

	transformacionConCompartida << <blocksPerGrid, threadsPerBlock, 2 * blocksize * sizeof(float) >> > (d_A, d_B, d_C, N);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch kernel! ERROR= %d\n", err);
		exit(EXIT_FAILURE);
	}

	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	double Tgpu = (clock() - t1) / CLOCKS_PER_SEC;

	cout << "Tiempo gastado GPU " << Tgpu << endl << endl;
}
