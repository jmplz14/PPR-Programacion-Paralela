#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string.h>
#include <time.h>


// CUDA runtime
//#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>




using namespace std;

__device__ void reduce(float * d_suma, float * sdata_suma, float * d_max, float * sdata_max, int tid, int i)
{
	
	/*int tid = threadIdx.x;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = ((i < N) ? d_V[i] : 0.0f);
	__syncthreads();*/

	for (int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
			sdata_suma[tid] += sdata_suma[tid + s];
			
			float valor = max(sdata_max[tid + s],sdata_max[tid]);
			sdata_max[tid] = valor;
			/*if( sdata_max[tid] < valor)
				sdata_max[tid] = valor;*/
		}
		
		__syncthreads();
	}

	if (tid == 0){
	 d_suma[blockIdx.x] = sdata_suma[0];
	 d_max[blockIdx.x] = sdata_max[0];
	}
	
}

__global__ void transformacionSinCompartida(float * A, float * B, float * C, float * D_suma, float * D_max, int N) {
	extern __shared__ float sdata[];
	int tid = threadIdx.x;
	int i = tid + blockDim.x * blockIdx.x;
	if (i < N) {

		float *sdata_suma = sdata;
		float *sdata_max = sdata + blockDim.x;
		

		int posInicio = blockIdx.x * blockDim.x;
		float suma = 0;
		for (int j = 0; j < blockDim.x; j++) {
			int posActual = posInicio + j;
			float valorA = A[posActual] * i;
			if ( (int)ceil(valorA) % 2 == 0) {
				suma += valorA + B[posActual];
			}
			else {
				suma += valorA - B[posActual];
			}
		}
		
		C[i] = suma;
		sdata_suma[tid] = suma;
		sdata_max[tid] = suma;
		
		__syncthreads();


		reduce(D_suma, sdata_suma, D_max, sdata_max, tid, i);
	}
	
}

__global__ void transformacionConCompartida(float * A, float * B, float * C, float * D_suma, float * D_max, int N) {
	extern __shared__ float sdata[];
	int tid = threadIdx.x;
	int i = tid + blockDim.x * blockIdx.x;
	
	float *sdata_A = sdata; 
	float *sdata_B = sdata + blockDim.x; 
	float *sdata_suma = sdata + blockDim.x*2;
	float *sdata_max = sdata + blockDim.x*3;

	sdata_A[tid] = A[i];
	sdata_B[tid] = B[i];
	
	__syncthreads();


	if (i < N) {


		float suma = 0;
		
		for (int j = 0; j < blockDim.x; j++) {
			float valorA = sdata_A[j] * i;
			if ( (int)ceil(valorA) % 2 == 0) {
				suma += valorA + sdata_B[j];
			}
			else {
				suma += valorA - sdata_B[j];
			}
		}

		
		C[i] = suma;
		sdata_suma[tid] = suma;
		sdata_max[tid] = suma;
		
		__syncthreads();

		
		reduce(D_suma, sdata_suma, D_max, sdata_max, tid, i);
		
		
		
	}

}








int main(int argc, char *argv[]) {

	int blocksize, NBlocks;
	if (argc != 3)
	{
		cout << "Uso: transformacion Num_bloques Tam_bloque  " << endl;
		return(0);
	}
	else
	{
		NBlocks = atoi(argv[1]);
		blocksize = atoi(argv[2]);
	}

	const int   N = blocksize * NBlocks;

	float *A = new float[N];
	float *B = new float[N];
	float *C = new float[N];
	float *D = new float[NBlocks];
	float *D_suma = new float[NBlocks];
	float *D_max = new float[NBlocks];
	float *D_suma_g = new float[NBlocks];
	float *D_max_g = new float[NBlocks];

	int devID;
	hipError_t err;
	err = hipGetDevice(&devID);
	if (err != hipSuccess) {
		cout << "ERRORRR" << endl;
	}
	
	int size = N * sizeof(float);
	float * d_A = NULL, *d_B = NULL, *d_C = NULL, *d_suma = NULL, *d_max = NULL, *d_suma_g = NULL, *d_max_g = NULL;

	err = hipMalloc((void **)&d_A, size);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA A" << endl;
	}

	err = hipMalloc((void **)&d_B, size);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA B" << endl;
	}

	err = hipMalloc((void **)&d_C, size);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA C" << endl;
	}
	err = hipMalloc((void **)&d_suma, NBlocks*sizeof(float));
	if (err != hipSuccess) {
		cout << "ERROR RESERVA suma compartida" << endl;
	}
	err = hipMalloc((void **)&d_max, NBlocks*sizeof(float));
	if (err != hipSuccess) {
		cout << "ERROR RESERVA maximo compartido" << endl;
	}
	err = hipMalloc((void **)&d_suma_g, NBlocks * sizeof(float));
	if (err != hipSuccess) {
		cout << "ERROR RESERVA suma glogal" << endl;
	}
	err = hipMalloc((void **)&d_max_g, NBlocks * sizeof(float));
	if (err != hipSuccess) {
		cout << "ERROR RESERVA max global" << endl;
	}

	
	for (int i = 0; i < N; i++)
	{
		/*A[i] = 1;
		B[i] = 2;*/
		A[i] = (float)(1 - (i % 100)*0.001);
		B[i] = (float)(0.5 + (i % 10) *0.1);
	}

	err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "ERROR COPIA A" << endl;
	}

	err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "ERROR COPIA B" << endl;
	}
	

//----------------------------------------Memoria compartida----------------------------------------------------------------
	double  t1 = clock();

	transformacionConCompartida << <NBlocks, blocksize, 4 * blocksize * sizeof(float) >> > (d_A, d_B, d_C, d_suma, d_max, N);
	
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch kernel! ERROR= %d\n", err);
		exit(EXIT_FAILURE);
	}
	hipMemcpy(D_max, d_max, NBlocks*sizeof(float), hipMemcpyDeviceToHost);
	
	hipMemcpy(D_suma, d_suma, NBlocks*sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	
	float mayor = D_max[0];
	for (int k = 1; k < NBlocks; k++) {
		if (D_max[k] > mayor)
			mayor = D_max[k];
	}
	double TgpuCompartida = (clock() - t1) / CLOCKS_PER_SEC;

	cout << "-----------------------GPU COMPARTIDA---------------------------------" << endl;
	cout << "Tiempo gastado GPU compartida: " << TgpuCompartida << endl << endl;
	cout << "El mayor es: " << mayor << endl;
//----------------------------------------Memoria Global----------------------------------------------------------------
	t1 = clock();

	transformacionSinCompartida << <NBlocks, blocksize, 2 * blocksize * sizeof(float) >> > (d_A, d_B, d_C, d_suma_g, d_max_g, N);

	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch kernel! ERROR= %d\n", err);
		exit(EXIT_FAILURE);
	}
	hipMemcpy(D_max_g, d_max_g, NBlocks * sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(D_suma_g, d_suma_g, NBlocks * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	
	float mayor_global = D_max_g[0];
	for (int k = 1; k < NBlocks; k++) {
		if (D_max_g[k] > mayor_global) 
			mayor_global = D_max_g[k];
		
	}
	
	double TgpuGlobal = (clock() - t1) / CLOCKS_PER_SEC;
	cout << "-----------------------GPU GLOBAL---------------------------------" << endl;
	cout << "Tiempo gastado GPU global: " << TgpuGlobal << endl << endl;
	cout << "El mayor es: " << mayor_global << endl;
//--------------------------------------------Secuencial----------------------------------------------------------------
t1=clock();

  
float mx; 
// Compute C[i], d[K] and mx
for (int k=0; k<NBlocks;k++)
{ int istart=k*blocksize;
  int iend  =istart+blocksize;
  D[k]=0.0;
  for (int i=istart; i<iend;i++)
  { C[i]=0.0;
    for (int j=istart; j<iend;j++)
     { float a=A[j]*i;
       if ((int)ceil(a) % 2 ==0)
	C[i]+= a + B[j];
       else
 	C[i]+= a - B[j];
     }
   D[k]+=C[i];
   mx=(i==1)?C[0]:max(C[i],mx);
  }
}

  double TSecuencial = (clock() - t1) / CLOCKS_PER_SEC;
  


  cout << "--------------------------Secunencial---------------------------------" << endl;
  cout << "Tiempo gastado Secuencial: " << TSecuencial << endl << endl;
  cout << "El mayor es: " << mx << endl;
}
